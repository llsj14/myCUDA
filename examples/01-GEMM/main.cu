#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <stdio.h>

template <typename T>
void allocate_matrix_memory(int row, int col, T** matrix) {
    hipError_t err = hipMalloc((void**)matrix, row * col * sizeof(T));
    if(hipSuccess != err) {
        printf("CUDA:ERROR: hipMalloc failure %s\n", hipGetErrorString(err));
        exit(1);
    }
}

template <typename T>
void set_matrix_random_number(int row, int col, T* matrix, hiprandGenerator_t generator) {
	if (sizeof(T) == sizeof(float)) {
		hiprandStatus_t status = hiprandGenerateUniform(generator, matrix, row * col);
		if (HIPRAND_STATUS_SUCCESS != status) {
			printf("CUDA:ERROR: curandGenerationUniform failure %d\n", status);
			exit(1);
		}
	}
	else {
		printf("set matrix random number only supports float now");
		exit(1);
	}
}

template <typename T>
void free_matrix_memory(T* matrix) {
    hipError_t err = hipFree(matrix);
    if(hipSuccess != err) {
        printf("CUDA:ERROR: hipFree failure %s\n", hipGetErrorString(err));
        exit(1);
    }
}

void test_cublas_sgemm_ex(int m, int n, int k) {
    float *A = nullptr;
    float *B = nullptr;
    float *C = nullptr;
    allocate_matrix_memory(m, k, &A);
    allocate_matrix_memory(n, k, &B);
    allocate_matrix_memory(m, n, &C);

    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator, 123456789ULL);
    set_matrix_random_number(m, k, A, generator);
    set_matrix_random_number(n, k, B, generator);

    const float alpha = 1.0;
    const float beta  = 0.0;
    static hipblasHandle_t handle = nullptr;
    hipblasCreate(&handle);
    
    hipDataType Atype, Btype, Ctype;
    Atype = Btype = Ctype= HIP_R_32F;
    
    cublasSgemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, A, Atype, m, B, Btype, n, &beta, C, Ctype, k);

    float *C_cpu = (float*)malloc(n * k);
    hipMemcpy(C_cpu, C, n * k, hipMemcpyDeviceToHost);
    for (int i=0; i<10; i++) {
        printf("%f\n", C_cpu[i]);
    }
    free(C_cpu);

    hiprandDestroyGenerator(generator);
    free_matrix_memory(A);
    free_matrix_memory(B);
    free_matrix_memory(C);
}

int main() {
    test_cublas_sgemm_ex(16, 16, 16);

    return 0;
}

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

template <typename T>
void allocate_matrix_memory(int row, int col, T* matrix) {
    hipError_t err = hipMalloc(&matrix, row * col * sizeof(T));
    if(hipSuccess != err) {
        printf("CUDA:ERROR: hipMalloc failure %s\n", hipGetErrorString(err));
        exit(1);
    }
}

template <typename T>
void free_matrix_memory(T* matrix) {
    hipError_t err = hipFree(matrix);
    if(hipSuccess != err) {
        printf("CUDA:ERROR: hipFree failure %s\n", hipGetErrorString(err));
        exit(1);
    }
}

void test_cublas_sgemm_ex(int m, int n, int k) {
    float *A = nullptr;
    float *B = nullptr;
    float *C = nullptr;
    allocate_matrix_memory(m, k, A);
    allocate_matrix_memory(n, k, B);
    allocate_matrix_memory(m, n, C);

    const float alpha = 1.0;
    const float beta  = 0.0;
    static hipblasHandle_t handle = nullptr;
    hipblasCreate(&handle);
    
    hipDataType Atype, Btype, Ctype;
    Atype = Btype = Ctype= HIP_R_32F;
    
    cublasSgemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, A, Atype, m, B, Btype, n, &beta, C, Ctype, k);
    
    free_matrix_memory(A);
    free_matrix_memory(B);
    free_matrix_memory(C);
}

int main() {
    test_cublas_sgemm_ex(16, 16, 16);

    return 0;
}
